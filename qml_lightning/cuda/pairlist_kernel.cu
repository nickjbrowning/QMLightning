#include "hip/hip_runtime.h"
#include<torch/torch.h>
#include<cmath>

using namespace std;

__device__ void get_pbc_dij(float *drij, float *cell_vectors, float *inv_cell_vectors) {

	/*
	 *   h := [a, b, c], a=(a1,a2,a3), ... (the matrix of box vectors)
	 r_ij := r_i - r_j                 (difference vector)

	 s_i = h^{-1} r_i
	 s_ij = s_i - s_j
	 s_ij <-- s_ij - NINT(s_ij)        (general minimum image convention)
	 r_ij = h s_ij
	 */

	for (int x = 0; x < 3; x++) {

		float sij_x = 0.0;
		float rij_x = 0.0;

		for (int y = 0; y < 3; y++) {
			sij_x += inv_cell_vectors[x * 3 + y] * drij[x];
		}

		sij_x = sij_x - round(sij_x);

		for (int y = 0; y < 3; y++) {
			rij_x += cell_vectors[x * 3 + y] * sij_x;
		}

		drij[x] = rij_x;
	}
}

__global__ void get_num_neighbours_kernel(const torch::PackedTensorAccessor32<float, 3, torch::RestrictPtrTraits> coordinates,
		const torch::PackedTensorAccessor32<int, 1, torch::RestrictPtrTraits> natom_counts, float rcut2,
		const torch::PackedTensorAccessor32<float, 3, torch::RestrictPtrTraits> lattice_vectors,
		const torch::PackedTensorAccessor32<float, 3, torch::RestrictPtrTraits> inv_lattice_vectors,
		torch::PackedTensorAccessor32<int, 2, torch::RestrictPtrTraits> num_neighbours) {

	extern __shared__ float s[];
	float *shared_x = (float*) &s;
	float *shared_y = (float*) &shared_x[blockDim.y];
	float *shared_z = (float*) &shared_y[blockDim.y];
	int *atomIDs = (int*) &shared_z[blockDim.y];
	float *slattice_vecs = (float*) &atomIDs[blockDim.y];
	float *sinv_lattice_vecs = (float*) &slattice_vecs[9];

	int batchID = blockIdx.x;
	int iatom = blockIdx.y * blockDim.y + threadIdx.y;

	int idy = threadIdx.y;

	int natoms = natom_counts[batchID];

	int ntiles = int(ceil(float(natoms) / blockDim.y));

	int num_neighbour_atoms_i = 0;

	float rix = -HUGE_VALF;
	float riy = -HUGE_VALF;
	float riz = -HUGE_VALF;

	bool pbc = false;

	float drij[3];

	if (lattice_vectors.size(0) > 0) {

		pbc = true;

		if (threadIdx.x < 3) {
			for (int j = 0; j < 3; j++) {
				slattice_vecs[threadIdx.x * 3 + j] = lattice_vectors[batchID][threadIdx.x][j];
				sinv_lattice_vecs[threadIdx.x * 3 + j] = inv_lattice_vectors[batchID][threadIdx.x][j];
			}
		}
	}

	__syncthreads();

	if (iatom < natoms) {
		rix = coordinates[batchID][iatom][0];
		riy = coordinates[batchID][iatom][1];
		riz = coordinates[batchID][iatom][2];
	}
//now loop through all atoms for batchID

	for (int tile = 0; tile < ntiles; tile++) {
		int jdx = tile * blockDim.y + threadIdx.y;

		if (jdx < natoms) {
			shared_x[idy] = coordinates[batchID][jdx][0];
			shared_y[idy] = coordinates[batchID][jdx][1];
			shared_z[idy] = coordinates[batchID][jdx][2];
			atomIDs[idy] = jdx;
		} else {
			shared_x[idy] = HUGE_VALF;
			shared_y[idy] = HUGE_VALF;
			shared_z[idy] = HUGE_VALF;
			atomIDs[idy] = -1;
		}

		__syncthreads();

		for (int j = 0; j < min(natoms, blockDim.y); j++) {

			if (atomIDs[j] == -1) {
				continue;
			}

			drij[0] = rix - shared_x[j];
			drij[1] = riy - shared_y[j];
			drij[2] = riz - shared_z[j];

			float rij2 = 0.0;

			if (pbc) {
				get_pbc_dij(drij, slattice_vecs, sinv_lattice_vecs);
			}

			rij2 = drij[0] * drij[0] + drij[1] * drij[1] + drij[2] * drij[2];

			if (rij2 < rcut2 && rij2 > 0 && iatom < natoms) {
				num_neighbour_atoms_i++;
			}
		}
	}

	if (iatom < natoms) {
		num_neighbours[batchID][iatom] = num_neighbour_atoms_i;
	}
}

__global__ void get_neighbour_list_kernel(const torch::PackedTensorAccessor32<float, 3, torch::RestrictPtrTraits> coordinates,
		const torch::PackedTensorAccessor32<int, 1, torch::RestrictPtrTraits> natom_counts, float rcut2,
		const torch::PackedTensorAccessor32<float, 3, torch::RestrictPtrTraits> lattice_vectors,
		const torch::PackedTensorAccessor32<float, 3, torch::RestrictPtrTraits> inv_lattice_vectors,
		torch::PackedTensorAccessor32<int, 3, torch::RestrictPtrTraits> neighbour_list) {

	extern __shared__ float s[];
	float *shared_x = (float*) &s;
	float *shared_y = (float*) &shared_x[blockDim.y];
	float *shared_z = (float*) &shared_y[blockDim.y];
	int *atomIDs = (int*) &shared_z[blockDim.y];
	float *slattice_vecs = (float*) &atomIDs[blockDim.y];
	float *sinv_lattice_vecs = (float*) &slattice_vecs[9];

	int batchID = blockIdx.x;
	int iatom = blockIdx.y * blockDim.y + threadIdx.y;

	int idy = threadIdx.y;

	int natoms = natom_counts[batchID];
	int count = 0;

	int ntiles = int(ceil(float(natoms) / blockDim.y));

	float rix = -HUGE_VALF;
	float riy = -HUGE_VALF;
	float riz = -HUGE_VALF;

	bool pbc = false;

	float drij[3];

	if (lattice_vectors.size(0) > 0) {

		pbc = true;

		if (threadIdx.x < 3) {
			for (int j = 0; j < 3; j++) {
				slattice_vecs[threadIdx.x * 3 + j] = lattice_vectors[batchID][threadIdx.x][j];
				sinv_lattice_vecs[threadIdx.x * 3 + j] = inv_lattice_vectors[batchID][threadIdx.x][j];
			}
		}
	}

	if (iatom < natoms) {
		rix = coordinates[batchID][iatom][0];
		riy = coordinates[batchID][iatom][1];
		riz = coordinates[batchID][iatom][2];
	}
//now loop through all atoms for batchID

	for (int tile = 0; tile < ntiles; tile++) {
		int jdx = tile * blockDim.y + threadIdx.y;

		if (jdx < natoms) {
			shared_x[idy] = coordinates[batchID][jdx][0];
			shared_y[idy] = coordinates[batchID][jdx][1];
			shared_z[idy] = coordinates[batchID][jdx][2];
			atomIDs[idy] = jdx;
		} else {
			shared_x[idy] = HUGE_VALF;
			shared_y[idy] = HUGE_VALF;
			shared_z[idy] = HUGE_VALF;
			atomIDs[idy] = -1;
		}

		__syncthreads();

		for (int j = 0; j < min(natoms, blockDim.y); j++) {

			int jidx = atomIDs[j];

			if (jidx == -1) {
				continue;
			}

			drij[0] = rix - shared_x[j];
			drij[1] = riy - shared_y[j];
			drij[2] = riz - shared_z[j];

			float rij2 = 0.0;

			if (pbc) {
				get_pbc_dij(drij, slattice_vecs, sinv_lattice_vecs);
			}

			rij2 = drij[0] * drij[0] + drij[1] * drij[1] + drij[2] * drij[2];

			if (rij2 < rcut2 && rij2 > 0) {
				neighbour_list[batchID][iatom][count] = jidx;
				count++;
			}
		}
	}

}

__global__ void safe_fill_kernel(torch::PackedTensorAccessor32<int, 3, torch::RestrictPtrTraits> pairlist) {

	int batch_num = pairlist.size(0);
	int natoms = pairlist.size(1);

	int batchID = blockIdx.x;
	int iatom = blockIdx.y * blockDim.y + threadIdx.y;

	if (iatom < natoms) {

		int newatm = -1;

		if (iatom == 0) {
			newatm = 1;
		} else {
			newatm = iatom - 1;
		}

		for (int k = 0; k < pairlist.size(2); k++) {
			int curval = pairlist[batchID][iatom][k];

			if (curval == -1) {
				pairlist[batchID][iatom][k] = newatm;
			}
		}
	}
}

void safeFillCUDA(torch::Tensor pairlist) {
	int currBatchSize = pairlist.size(0);
	int natoms = pairlist.size(1);

	const int nthreads = 64;

	dim3 numBlocks(currBatchSize, int(ceil(float(natoms) / nthreads)));
	dim3 threadsPerBlock(1, nthreads);

safe_fill_kernel<<<numBlocks, threadsPerBlock>>>( pairlist.packed_accessor32<int, 3, torch::RestrictPtrTraits>());

}

void getNumNeighboursCUDA(torch::Tensor coordinates, torch::Tensor natom_counts, float rcut, torch::Tensor lattice_vecs, torch::Tensor inv_lattice_vecs,
	torch::Tensor num_neighbours) {

const int nthreads = 64;

int currBatchSize = coordinates.size(0);
int natoms = natom_counts.max().item<int>();

int nBlockY = int(ceil(float(natoms) / nthreads));

float rcut2 = rcut * rcut;

dim3 numBlocks(currBatchSize, nBlockY);
dim3 threadsPerBlock(1, nthreads);

//printf("natoms: %d BlockX: %d nBlockY: %d\n", natoms, currBatchSize, nBlockY);

get_num_neighbours_kernel<<<numBlocks, threadsPerBlock, (4 * nthreads + 18) * sizeof(float)>>>(
		coordinates.packed_accessor32<float, 3, torch::RestrictPtrTraits>(),
		natom_counts.packed_accessor32<int, 1, torch::RestrictPtrTraits>(),
		rcut2,
		lattice_vecs.packed_accessor32<float, 3, torch::RestrictPtrTraits>(),
		inv_lattice_vecs.packed_accessor32<float, 3, torch::RestrictPtrTraits>(),
		num_neighbours.packed_accessor32<int, 2, torch::RestrictPtrTraits>());

hipDeviceSynchronize();

}

void getNeighbourListCUDA(torch::Tensor coordinates, torch::Tensor natom_counts, float rcut, torch::Tensor lattice_vecs, torch::Tensor inv_lattice_vecs,
	torch::Tensor neighbour_list) {

int currBatchSize = coordinates.size(0);
int natoms = natom_counts.max().item<int>();

float rcut2 = rcut * rcut;

const int nthreads = 64;

dim3 numBlocks(currBatchSize, int(ceil(float(natoms) / nthreads)));
dim3 threadsPerBlock(1, nthreads);

get_neighbour_list_kernel<<<numBlocks, threadsPerBlock, (4 * nthreads + 18) * sizeof(float)>>>(
		coordinates.packed_accessor32<float, 3, torch::RestrictPtrTraits>(),
		natom_counts.packed_accessor32<int, 1, torch::RestrictPtrTraits>(),
		rcut2,
		lattice_vecs.packed_accessor32<float, 3, torch::RestrictPtrTraits>(),
		inv_lattice_vecs.packed_accessor32<float, 3, torch::RestrictPtrTraits>(),
		neighbour_list.packed_accessor32<int, 3, torch::RestrictPtrTraits>());

hipDeviceSynchronize();
}

