#include "hip/hip_runtime.h"
#include<torch/torch.h>

using namespace std;

__global__ void hadamard_kernel(const torch::PackedTensorAccessor32<float, 3, torch::RestrictPtrTraits> input,
		torch::PackedTensorAccessor32<float, 3, torch::RestrictPtrTraits> output, int log2N) {

	const int N = 1 << log2N;

	extern __shared__ float s[];

	const float normh = (1.0 / powf(2.0, float(log2N) / 2));
	const int nstacks = input.size(1);

	for (int stack = 0; stack < nstacks; stack++) {

		for (int pos = threadIdx.x; pos < N; pos += blockDim.x) {
			s[pos] = input[blockIdx.x][stack][pos];
		}

		/**Hadamard transform taken from Nvidia Cuda Examples**/

		int stride = 1;

		//Do single radix-2 stage for odd power of two
		if (log2N & 1) {

			__syncthreads();

			for (int pos = threadIdx.x; pos < N / 2; pos += blockDim.x) {
				int i0 = pos << 1;
				int i1 = i0 + 1;

				float D0 = s[i0];
				float D1 = s[i1];
				s[i0] = D0 + D1;
				s[i1] = D0 - D1;
			}
			stride <<= 1;
		}

		//Main radix-4 stages
		const int pos = threadIdx.x;

		for (; stride <= N >> 2; stride <<= 2) {
			int lo = pos & (stride - 1);
			int i0 = ((pos - lo) << 2) + lo;
			int i1 = i0 + stride;
			int i2 = i1 + stride;
			int i3 = i2 + stride;

			__syncthreads();

			float D0 = s[i0];
			float D1 = s[i1];
			float D2 = s[i2];
			float D3 = s[i3];

			float T;
			T = D0;
			D0 = D0 + D2;
			D2 = T - D2;
			T = D1;
			D1 = D1 + D3;
			D3 = T - D3;
			T = D0;
			s[i0] = D0 + D1;
			s[i1] = T - D1;
			T = D2;
			s[i2] = D2 + D3;
			s[i3] = T - D3;
		}

		__syncthreads();

		/**Finished Hadamard transform for subblock N/d.*/

		//normalize hadamard transform
		for (int pos = threadIdx.x; pos < N; pos += blockDim.x) {
			s[pos] = normh * s[pos];
		}

		__syncthreads();

		//save [HD]n stack to global memory
		for (int pos = threadIdx.x; pos < N; pos += blockDim.x) {
			output[blockIdx.x][stack][pos] = s[pos];
		}
	}
}

__global__ void sorf_matrix_kernel_float(const torch::PackedTensorAccessor32<float, 2, torch::RestrictPtrTraits> input,
		const torch::PackedTensorAccessor32<float, 3, torch::RestrictPtrTraits> D, torch::PackedTensorAccessor32<float, 2, torch::RestrictPtrTraits> output,
		int nstacks, int log2N) {

	/**
	 * Computes the structured orthogonal matrix W from [HD]_n, where D is a rademacher-distributed diagonal matrix
	 * and H is the Hadamard matrix. n corresponds to the number of [HD] operations to perform.
	 *
	 * input is the [natoms, repsize] representation matrix. This should be subselected from the full representation based on element types
	 * such that each element type is transformed in the same way via element-specific D's.
	 *
	 * output is the [natoms, nfeatures] dot product matrix [Wx], where each column of W has been stacked N/d times.
	 *
	 * D is the [n, nstacks, d] rademacher tensor.
	 *
	 * **/

	const int N = 1 << log2N;

	extern __shared__ float s[];

	//if (blockIdx.x == 0 && threadIdx.x == 0)
	//	printf("check: %d %f", N, powf(2.0, float(log2N) / 2));

	int mdiag = D.size(0); // number of [HD] blocks to compute
	//loop over N/d hadamard transforms to create length-N feature vector

	const float normh = (1.0 / powf(2.0, float(log2N) / 2));

	for (int stack = 0; stack < nstacks; stack++) {

		for (int pos = threadIdx.x; pos < N; pos += blockDim.x) {
			s[pos] = input[blockIdx.x][pos];
		}

		//loop over n [HD] blocks
		for (int m = 0; m < mdiag; m++) {

			for (int pos = threadIdx.x; pos < N; pos += blockDim.x) {
				s[pos] = D[m][stack][pos] * s[pos];
			}

			/**Hadamard transform taken from Nvidia Cuda Examples**/

			int stride = 1;

			//Do single radix-2 stage for odd power of two
			if (log2N & 1) {

				__syncthreads();

				for (int pos = threadIdx.x; pos < N / 2; pos += blockDim.x) {
					int i0 = pos << 1;
					int i1 = i0 + 1;

					float D0 = s[i0];
					float D1 = s[i1];
					s[i0] = D0 + D1;
					s[i1] = D0 - D1;
				}
				stride <<= 1;
			}

			//Main radix-4 stages
			const int pos = threadIdx.x;

			for (; stride <= N >> 2; stride <<= 2) {
				int lo = pos & (stride - 1);
				int i0 = ((pos - lo) << 2) + lo;
				int i1 = i0 + stride;
				int i2 = i1 + stride;
				int i3 = i2 + stride;

				__syncthreads();

				float D0 = s[i0];
				float D1 = s[i1];
				float D2 = s[i2];
				float D3 = s[i3];

				float T;
				T = D0;
				D0 = D0 + D2;
				D2 = T - D2;
				T = D1;
				D1 = D1 + D3;
				D3 = T - D3;
				T = D0;
				s[i0] = D0 + D1;
				s[i1] = T - D1;
				T = D2;
				s[i2] = D2 + D3;
				s[i3] = T - D3;
			}

			__syncthreads();

			/**Finished Hadamard transform for subblock N/d.*/

			//normalize hadamard transform
			for (int pos = threadIdx.x; pos < N; pos += blockDim.x) {
				s[pos] = normh * s[pos];
			}
		}

		__syncthreads();

		//save [HD]n stack to global memory
		for (int pos = threadIdx.x; pos < N; pos += blockDim.x) {
			output[blockIdx.x][stack * N + pos] = s[pos];
		}
	}
}

__global__ void compute_featurisation_float(const torch::PackedTensorAccessor32<float, 2, torch::RestrictPtrTraits> coefficients,
		const torch::PackedTensorAccessor32<float, 1, torch::RestrictPtrTraits> bias,
		const torch::PackedTensorAccessor32<int, 1, torch::RestrictPtrTraits> ordering,
		torch::PackedTensorAccessor32<float, 2, torch::RestrictPtrTraits> features) {

	//coefficients: natoms, nfeatures
	//features nbatch, nfeatures
	//ordering: contains the indexes of which nbatch to add atom j to.

	int nfeatures = coefficients.size(1);
	int natoms = coefficients.size(0);

	int iatom = blockIdx.x;

	int batchID = ordering[iatom];

	const float normf = sqrt(2.0 / float(nfeatures));

	for (int N = threadIdx.x; N < nfeatures; N += blockDim.x) {
		atomicAdd(&features[batchID][N], cos(coefficients[iatom][N] + bias[N]) * normf);
	}
}

__global__ void precompute_sin_coeffs(const torch::PackedTensorAccessor32<float, 2, torch::RestrictPtrTraits> coefficients,
		const torch::PackedTensorAccessor32<float, 1, torch::RestrictPtrTraits> bias,
		torch::PackedTensorAccessor32<float, 2, torch::RestrictPtrTraits> output) {

	/**
	 * precompute the derivative of the features to save some time for the full derivative
	 *
	 * **/

	int nfeatures = coefficients.size(1);
	int natoms = coefficients.size(0);

	int iatom = blockIdx.x;

	const float normf = sqrt(2.0 / float(nfeatures));

	for (int N = threadIdx.x; N < nfeatures; N += blockDim.x) {
		output[iatom][N] = -sin(coefficients[iatom][N] + bias[N]) * normf;
	}
}

__global__ void compute_featurisation_derivative_float(const torch::PackedTensorAccessor32<float, 2, torch::RestrictPtrTraits> cos_derivs,
		const torch::PackedTensorAccessor32<int, 1, torch::RestrictPtrTraits> ordering,
		const torch::PackedTensorAccessor32<float, 4, torch::RestrictPtrTraits> input_derivative,
		const torch::PackedTensorAccessor32<float, 3, torch::RestrictPtrTraits> D, int nstacks, int log2N,
		torch::PackedTensorAccessor32<float, 4, torch::RestrictPtrTraits> feature_derivatives) {

	const int N = 1 << log2N;

	int nfeatures = cos_derivs.size(1);

	extern __shared__ float s[];

	float *u = (float*) &s;
	float *load_u = (float*) &u[N];

	int mdiag = D.size(0); // number of [HD] blocks to compute
//loop over N/d hadamard transforms to create length-N feature vector

	int nderiv_atoms = input_derivative.size(1);

	int iatom = int(floor(float(blockIdx.x) / nderiv_atoms));
	int jatom = blockIdx.x % nderiv_atoms;

	int batchID = ordering[iatom];

	const float normc = (1.0 / powf(2.0, float(log2N) / 2.0));
	const float feature_norm = sqrt(2.0 / float(nfeatures));

//printf("thread %d block %d iatom %d jatom %d batchID %d nstacks %d\n", threadIdx.x, blockIdx.x, iatom, jatom, batchID, nstacks);
	for (int x = 0; x < 3; x++) {

		for (int pos = threadIdx.x; pos < N; pos += blockDim.x) {
			load_u[pos] = input_derivative[iatom][jatom][x][pos];
		}

		for (int stack = 0; stack < nstacks; stack++) {

			for (int pos = threadIdx.x; pos < N; pos += blockDim.x) {
				u[pos] = load_u[pos];
			}

			__syncthreads();

			//loop over n [HD] blocks
			for (int m = 0; m < mdiag; m++) {

				for (int pos = threadIdx.x; pos < N; pos += blockDim.x) {
					u[pos] = D[m][stack][pos] * u[pos];
				}

				__syncthreads();

				/**Hadamard transform taken from Nvidia Cuda Examples**/

				int stride = 1;

				//Do single radix-2 stage for odd power of two
				if (log2N & 1) {

					__syncthreads();

					for (int pos = threadIdx.x; pos < N / 2; pos += blockDim.x) {
						int i0 = pos << 1;
						int i1 = i0 + 1;

						float D0 = u[i0];
						float D1 = u[i1];
						u[i0] = D0 + D1;
						u[i1] = D0 - D1;
					}
					stride <<= 1;
				}

				//Main radix-4 stages
				const int pos = threadIdx.x;

				for (; stride <= N >> 2; stride <<= 2) {
					int lo = pos & (stride - 1);
					int i0 = ((pos - lo) << 2) + lo;
					int i1 = i0 + stride;
					int i2 = i1 + stride;
					int i3 = i2 + stride;

					__syncthreads();

					float D0 = u[i0];
					float D1 = u[i1];
					float D2 = u[i2];
					float D3 = u[i3];

					float T;
					T = D0;
					D0 = D0 + D2;
					D2 = T - D2;
					T = D1;
					D1 = D1 + D3;
					D3 = T - D3;
					T = D0;

					u[i0] = D0 + D1;
					u[i1] = T - D1;
					T = D2;
					u[i2] = D2 + D3;
					u[i3] = T - D3;
				}

				__syncthreads();

				/**Finished Hadamard transform for subblock N/d.*/

				//normalize hadamard transform
				for (int pos = threadIdx.x; pos < N; pos += blockDim.x) {
					u[pos] = normc * u[pos];
				}
			}

			__syncthreads();

			//save d/dr cos([(HD)n] x + b)  stack to global memory
			for (int pos = threadIdx.x; pos < N; pos += blockDim.x) {

				int idx = stack * N + pos;

				float val = cos_derivs[iatom][idx] * u[pos];

				atomicAdd(&feature_derivatives[batchID][jatom][x][idx], val);

			}
		}
	}
}

__global__ void predict_forces_kernel(const torch::PackedTensorAccessor32<float, 2, torch::RestrictPtrTraits> cos_derivs,
		const torch::PackedTensorAccessor32<int, 1, torch::RestrictPtrTraits> ordering,
		const torch::PackedTensorAccessor32<float, 4, torch::RestrictPtrTraits> input_derivative,
		const torch::PackedTensorAccessor32<float, 3, torch::RestrictPtrTraits> D, int nstacks, int log2N,
		torch::PackedTensorAccessor32<float, 3, torch::RestrictPtrTraits> forces) {

//TODO
}

void hadamard_gpu(torch::Tensor input, torch::Tensor output) {

	int n = input.size(2);
	int log2N = int(log2(n));

	int curBatchSize = input.size(0);

	TORCH_CHECK(n == 1 << log2N, "input size must be power of 2.");

	hadamard_kernel<<<curBatchSize, (n+3)/4, n * sizeof(float)>>>(input.packed_accessor32<float, 3, torch::RestrictPtrTraits>(),
			output.packed_accessor32<float, 3, torch::RestrictPtrTraits>(), log2N);

	hipDeviceSynchronize();
}

void compute_sorf_matrix_gpu_float(torch::Tensor representations, torch::Tensor scaling, torch::Tensor sorf_matrix) {

	int n = representations.size(1);
	int log2N = int(log2(n));

	int curBatchSize = representations.size(0);

	int nfeatures = sorf_matrix.size(1);

	int log2f = int(log2(nfeatures));

	TORCH_CHECK(n == 1 << log2N, "representation size must be power of 2.");
	TORCH_CHECK(nfeatures == 1 << log2f, "features size must be power of 2.");

	int nstacks = int(float(nfeatures) / n);

	sorf_matrix_kernel_float<<<curBatchSize, (n+3)/4, n * sizeof(float)>>>(representations.packed_accessor32<float, 2, torch::RestrictPtrTraits>(),
			scaling.packed_accessor32<float, 3, torch::RestrictPtrTraits>(),
			sorf_matrix.packed_accessor32<float, 2, torch::RestrictPtrTraits>(), nstacks, log2N);

	hipDeviceSynchronize();
}

void compute_partial_feature_derivatives_gpu_float(torch::Tensor sorf_matrix, torch::Tensor bias, torch::Tensor sin_coeffs) {
	int currBatchSize = sorf_matrix.size(0);
	const int nthreads = 128;

	precompute_sin_coeffs<<<currBatchSize, nthreads>>>(
			sorf_matrix.packed_accessor32<float, 2, torch::RestrictPtrTraits>(),
			bias.packed_accessor32<float, 1, torch::RestrictPtrTraits>(),
			sin_coeffs.packed_accessor32<float, 2, torch::RestrictPtrTraits>());

	hipDeviceSynchronize();

}

void compute_molecular_featurization_gpu_float(torch::Tensor sorf_matrix, torch::Tensor bias, torch::Tensor ordering, torch::Tensor features) {

	int currBatchSize = sorf_matrix.size(0);
	const int nthreads = 128;

	compute_featurisation_float<<<currBatchSize, nthreads>>>(
			sorf_matrix.packed_accessor32<float, 2, torch::RestrictPtrTraits>(),
			bias.packed_accessor32<float, 1, torch::RestrictPtrTraits>(),
			ordering.packed_accessor32<int, 1, torch::RestrictPtrTraits>(),
			features.packed_accessor32<float, 2, torch::RestrictPtrTraits>());

	hipDeviceSynchronize();
}

void compute_molecular_featurization_derivative_gpu_float(torch::Tensor cos_derivs, torch::Tensor scaling, torch::Tensor input_derivatives,
		torch::Tensor ordering, torch::Tensor feature_derivatives) {

	int n = input_derivatives.size(3);
	int log2N = int(log2(n));

	int currBatchSize = input_derivatives.size(0) * input_derivatives.size(1);
//int currBatchSize = input_derivatives.size(0);
	int nfeatures = cos_derivs.size(1);

	int log2f = int(log2(nfeatures));

	TORCH_CHECK(n == 1 << log2N, "input_derivatives size must be power of 2.");
	TORCH_CHECK(nfeatures == 1 << log2f, "features size must be power of 2.");

	int nstacks = int(float(nfeatures) / n);

	int nthreads = (n + 3) / 4;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);

	compute_featurisation_derivative_float<<<currBatchSize, nthreads, 2*n * sizeof(float)>>>(
			cos_derivs.packed_accessor32<float, 2, torch::RestrictPtrTraits>(),
			ordering.packed_accessor32<int, 1, torch::RestrictPtrTraits>(),
			input_derivatives.packed_accessor32<float, 4, torch::RestrictPtrTraits>(),
			scaling.packed_accessor32<float, 3, torch::RestrictPtrTraits>(),
			nstacks, log2N,
			feature_derivatives.packed_accessor32<float, 4, torch::RestrictPtrTraits>());

	hipDeviceSynchronize();

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

//cout << "c+++ derivatives call time: " << milliseconds << endl;
}

